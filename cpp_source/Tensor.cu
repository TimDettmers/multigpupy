#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <vector>

//this looks strange, but otherwise we cannot use template classes together with export "C" for ctypes

template <typename T>
class TensorTemplate
{
public:
	  int batches;
	  int maps;
	  int rows;
	  int cols;
	  int size;
	  size_t bytes;
	  T *data;
	  int isCUDA;
	  int splitAxis;
	  std::vector<int*> shape_gpus;
	  std::vector<int> size_gpus;
	  std::vector<size_t> bytes_gpus;
	  std::vector<T*> data_gpus;

void freeTensor()
{
	if(isCUDA)
	{
		for(int i = 0;i < data_gpus.size(); i++)
		{
			hipSetDevice(i);
			hipFree(data_gpus[i]);
		}
		hipSetDevice(0);
	}
	else{ free(data); }
	free(this);
}



};
