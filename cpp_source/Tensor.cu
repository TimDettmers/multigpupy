#include <Tensor.cuh>

void Tensor::freeTensor()
{
	if(onGPU)
	{
		for(int i = 0;i < data_gpus.size(); i++)
		{
			CUDA_CHECK_RETURN(hipSetDevice(i));
			CUDA_CHECK_RETURN(hipFree(data_gpus[i]));
		}
		CUDA_CHECK_RETURN(hipSetDevice(0));
	}
	else{ free(data); }
	free(this);
}
