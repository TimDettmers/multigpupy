#include <basics.cuh>



Tensor *empty(int batches, int maps, int rows, int cols)
{
	Tensor *out = new Tensor();
	int size = batches*maps*rows*cols;
	size_t bytes = size*sizeof(float);
	out->batches = batches;
	out->maps = maps;
	out->rows = rows;
	out->cols = cols;
	out->bytes = bytes;
	out->size = size;
	out->onGPU = 1;

	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		float *gpu_data;
		CUDA_CHECK_RETURN(hipMalloc((void**)&gpu_data, bytes));

		if(i == 0){ out->data = gpu_data; }
		out->data_gpus.push_back(gpu_data);
	}

	CUDA_CHECK_RETURN(hipSetDevice(0));

	return out;
}

Tensor *zeros(int batches, int maps, int rows, int cols)
{
	Tensor *out = empty(batches,maps,rows,cols);
	return fill_with_number(out, 0.0f);
}

Tensor *ones(int batches, int maps, int rows, int cols)
{
	Tensor *out = empty(batches,maps,rows,cols);
	return fill_with_number(out, 1.0f);
}

Tensor *fill_with_number(Tensor *A, float fill_value)
{
	thrust::device_ptr<float> ptr_dev(A->data);
	thrust::fill(ptr_dev, ptr_dev + A->size,fill_value);
	return A;
}



Tensor *T(Tensor *A)
{
	Tensor *out = empty(A->batches,A->maps,A->cols,A->rows);
	T(A,out, A->rows,A->cols);
	out->rows = A->cols;
	out->cols = A->rows;
	return out;
}


void T(Tensor *A, Tensor *out,  int rows, int cols)
{

	 // setup execution parameters
	  int grid_x = rows / COPY_BLOCK_SIZE;
	  if (rows  % COPY_BLOCK_SIZE)
	    grid_x++;

	  int grid_y = cols / COPY_BLOCK_SIZE;
	  if (cols % COPY_BLOCK_SIZE)
	    grid_y++;


	  dim3 grid(grid_x, grid_y, A->maps);
	  dim3 threads(COPY_BLOCK_SIZE, COPY_BLOCK_SIZE, 1);
	  kTransposeTensor<<< grid, threads >>>(A->data, out->data, A->batches, rows, cols);
}

Tensor *to_col_major(Tensor *A)
{
  Tensor *out = empty(A->batches,A->maps,A->rows,A->cols);
  T(A, out, A->cols,A->rows);

  return out;
}

void to_col_major(Tensor *A, Tensor *out)
{
	T(A, out, A->cols,A->rows);
}

Tensor *to_row_major(Tensor *A)
{
	Tensor *out = empty(A->batches,A->maps,A->rows,A->cols);
	T(A, out, A->rows,A->cols);

  return out;
}



Tensor *tocpu(Tensor *A, float *cpu_buffer)
{
	Tensor *temp = to_row_major(A);
	Tensor *out = new Tensor();

	CUDA_CHECK_RETURN(hipMemcpy(cpu_buffer,temp->data,temp->bytes,hipMemcpyDefault));
	out->batches = temp->batches;
	out->maps = temp->maps;
	out->rows = temp->rows;
	out->cols = temp->cols;
	out->bytes = temp->bytes;
	out->size = temp->size;
	out->data = cpu_buffer;
	out->onGPU = 0;

	CUDA_CHECK_RETURN(hipFree(temp->data));
	delete temp;


	return out;
}

void togpu(Tensor *out, float *cpu_buffer)
{
	Tensor *temp = empty(out->batches,out->maps,out->rows,out->cols);
	CUDA_CHECK_RETURN(hipMemcpy(out->data,cpu_buffer,out->bytes,hipMemcpyDefault));
	to_col_major(out,temp);
	CUDA_CHECK_RETURN(hipMemcpy(out->data,temp->data,out->bytes,hipMemcpyDefault));
	CUDA_CHECK_RETURN(hipFree(temp->data));
	free(temp);
}


void add(Tensor *A, Tensor *B, Tensor *out)
{
	//checkMatrixOperation(A, B, out, HIPBLAS_OP_N, HIPBLAS_OP_N, 0);
	int block_size = (A->size/THREADS_PER_BLOCKS) + 1;
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kAdd<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->size);
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}


Tensor *add(Tensor *A, Tensor *B)
{
	Tensor *out = empty(A->batches,A->maps,A->rows,A->cols);
	add(A, B, out);

	return out;
}

void sub(Tensor *A, Tensor *B, Tensor *out)
{
	//checkMatrixOperation(A, B, out, HIPBLAS_OP_N, HIPBLAS_OP_N, 0);
	int block_size = (A->size/THREADS_PER_BLOCKS) + 1;
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kSub<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->size);
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}


Tensor *sub(Tensor *A, Tensor *B)
{
	Tensor *out = empty(A->batches,A->maps,A->rows,A->cols);
	sub(A, B, out);

	return out;
}

void mul(Tensor *A, Tensor *B, Tensor *out)
{
	//checkMatrixOperation(A, B, out, HIPBLAS_OP_N, HIPBLAS_OP_N, 0);
	int block_size = (A->size/THREADS_PER_BLOCKS) + 1;
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kMul<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->size);
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}


Tensor *mul(Tensor *A, Tensor *B)
{
	Tensor *out = empty(A->batches,A->maps,A->rows,A->cols);
	mul(A, B, out);

	return out;
}

void div(Tensor *A, Tensor *B, Tensor *out)
{
	//checkMatrixOperation(A, B, out, HIPBLAS_OP_N, HIPBLAS_OP_N, 0);
	int block_size = (A->size/THREADS_PER_BLOCKS) + 1;
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kDiv<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->size);
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}


Tensor *div(Tensor *A, Tensor *B)
{
	Tensor *out = empty(A->batches,A->maps,A->rows,A->cols);
	div(A, B, out);

	return out;
}


