#include <basics.cuh>

Slice *emptySlice()
{
	Slice *out = new Slice();
	out->batch_start = 0;
	out->batch_stop = INT_MAX;
	out->map_start = 0;
	out->map_stop = INT_MAX;
	out->row_start = 0;
	out->row_stop = INT_MAX;
	out->col_start = 0;
	out->col_stop = INT_MAX;

	return out;
}



Tensor *empty(int batches, int maps, int rows, int cols)
{

	Tensor *out = new Tensor();
	int size = batches*maps*rows*cols;
	size_t bytes = size*sizeof(float);
	out->batches = batches;
	out->maps = maps;
	out->rows = rows;
	out->cols = cols;
	out->bytes = bytes;
	out->size = size;
	out->isCUDA = 1;

	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		float *gpu_data;
		CUDA_CHECK_RETURN(hipMalloc((void**)&gpu_data, bytes));

		if(i == 0){ out->data = gpu_data; }
		out->data_gpus.push_back(gpu_data);
	}

	CUDA_CHECK_RETURN(hipSetDevice(0));

	return out;
}

Tensor *empty_pinned(int batches, int maps, int rows, int cols, float *cpu_buffer)
{
	Tensor *out = new Tensor();
	int size = batches*maps*rows*cols;
	float *pinned_data;
	size_t bytes = size*sizeof(float);
	CUDA_CHECK_RETURN(hipHostAlloc(&pinned_data, bytes, hipHostMallocPortable));
	if(cpu_buffer)
		CUDA_CHECK_RETURN(hipMemcpy(pinned_data,cpu_buffer,bytes,hipMemcpyDefault));
	out->batches = batches;
	out->maps = maps;
	out->rows = rows;
	out->cols = cols;
	out->bytes = bytes;
	out->size = size;
	out->isCUDA = 1;
	out->data = pinned_data;

	return out;
}

Tensor *zeros(int batches, int maps, int rows, int cols)
{
	Tensor *out = empty(batches,maps,rows,cols);
	return fill_with_number(out, 0.0f);
}

Tensor *ones(int batches, int maps, int rows, int cols)
{
	Tensor *out = empty(batches,maps,rows,cols);
	return fill_with_number(out, 1.0f);
}

Tensor *fill_with_number(Tensor *A, float fill_value)
{
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		thrust::device_ptr<float> ptr_dev(A->data_gpus[i]);
		thrust::fill(ptr_dev, ptr_dev + A->size,fill_value);
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));

	return A;
}



Tensor *T(Tensor *A)
{
	Tensor *out = empty(A->batches,A->maps,A->cols,A->rows);
	T(A,out, A->rows,A->cols);
	out->rows = A->cols;
	out->cols = A->rows;
	return out;
}


void T(Tensor *A, Tensor *out,  int rows, int cols)
{
	// setup execution parameters
	int grid_x = rows / COPY_BLOCK_SIZE;
	if (rows  % COPY_BLOCK_SIZE)
		grid_x++;

	int grid_y = cols / COPY_BLOCK_SIZE;
	if (cols % COPY_BLOCK_SIZE)
		grid_y++;

	dim3 grid(grid_x, grid_y, A->maps);
	dim3 threads(COPY_BLOCK_SIZE, COPY_BLOCK_SIZE, 1);
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kTransposeTensor<<< grid, threads >>>(A->data_gpus[i], out->data_gpus[i], A->batches, rows, cols);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}

Tensor *to_col_major(Tensor *A)
{
  Tensor *out = empty(A->batches,A->maps,A->rows,A->cols);
  T(A, out, A->cols,A->rows);

  return out;
}

void to_col_major(Tensor *A, Tensor *out)
{
	T(A, out, A->cols,A->rows);
}

Tensor *to_row_major(Tensor *A)
{
	Tensor *out = empty(A->batches,A->maps,A->rows,A->cols);
	T(A, out, A->rows,A->cols);

  return out;
}



Tensor *tocpu(Tensor *A, float *cpu_buffer)
{
	Tensor *temp = to_row_major(A);
	Tensor *out = new Tensor();

	CUDA_CHECK_RETURN(hipMemcpy(cpu_buffer,temp->data,temp->bytes,hipMemcpyDefault));
	out->batches = temp->batches;
	out->maps = temp->maps;
	out->rows = temp->rows;
	out->cols = temp->cols;
	out->bytes = temp->bytes;
	out->size = temp->size;
	out->data = cpu_buffer;
	out->isCUDA = 0;

	CUDA_CHECK_RETURN(hipFree(temp->data));
	delete temp;


	return out;
}

void togpu(Tensor *out, float *cpu_buffer)
{

	Tensor *temp = empty(out->batches,out->maps,out->rows,out->cols);
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++){ CUDA_CHECK_RETURN(hipMemcpy(out->data_gpus[i],cpu_buffer,out->bytes,hipMemcpyDefault)); }
	to_col_major(out,temp);
	for(int i = 0; i < gpus; i++){ CUDA_CHECK_RETURN(hipMemcpy(out->data_gpus[i],temp->data_gpus[i],out->bytes,hipMemcpyDefault)); }

	temp->freeTensor();
}


Tensor *applySliceFunc(Tensor *A, Slice *S)
{
	Tensor *out = zeros(S->batch_stop-S->batch_start,
						S->map_stop-S->map_start,
						S->row_stop-S->row_start,
						S->col_stop-S->col_start);

	applySliceFunc(A, S, out);

	return out;

}

void applySliceFunc(Tensor *A, Slice *S, Tensor *out)
{
	int block_size = (A->rows*A->cols/THREADS_PER_BLOCKS) + 1;
	dim3 grid(block_size, A->maps,A->batches);
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kSlice<<<dim3(A->batches, A->maps,1),dim3(32,32,1)>>>(A->data_gpus[i],out->data_gpus[i],
				S->batch_start, S->batch_stop,
				S->map_start, S->map_stop,
				S->row_start, S->row_stop,
				S->col_start, S->col_stop,
				A->rows,A->cols,out->batches,out->maps,out->cols,out->rows);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}

	CUDA_CHECK_RETURN(hipSetDevice(0));


}

Tensor *applyFunc(Tensor *A, Tensor *B, Operation_t ops){ return applyFunc(A,B,0.0f,ops); }
Tensor *applyFunc(Tensor *A, Tensor *B, float flt, Operation_t ops)
{
	Tensor *out = empty(A->batches,A->maps,A->rows,A->cols);
	applyFunc(A, B, out, flt, ops);

	return out;
}

void applyFunc(Tensor *A, Tensor *B, Tensor *out, Operation_t ops){ applyFunc(A,B,out,0.0f,ops); }
void applyFunc(Tensor *A, Tensor *B, Tensor *out, float flt, Operation_t ops)
{
	int block_size = (A->rows*A->cols/THREADS_PER_BLOCKS) + 1;
	dim3 grid(block_size, A->maps,A->batches);
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		switch(ops)
		{
			case copy: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size, flt, copy); break;
			case add_scalar: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size, flt, add_scalar); break;
			case mul_scalar: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size, flt, mul_scalar); break;
			case add_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->size, flt, add_tensor); break;
			case sub_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->size, flt, sub_tensor); break;
			case mul_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->size, flt, mul_tensor); break;
			case div_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->size, flt, div_tensor); break;
			case abs_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size, flt, abs_tensor); break;
			case log_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size, flt, log_tensor); break;
			case exp_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size, flt, exp_tensor); break;
			case pow_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size, flt, pow_tensor); break;
			case logistic: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size, flt, logistic); break;
			case logistic_grad: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size, flt, logistic_grad); break;
			case rectified_linear: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size, flt, rectified_linear); break;
			case eq_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i],A->size, flt,eq_tensor); break;
			case lt_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i],A->size, flt,lt_tensor); break;
			case gt_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i],A->size, flt,gt_tensor); break;
			case ge_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i],A->size, flt,ge_tensor); break;
			case le_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i],A->size, flt,le_tensor); break;
			case ne_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i],A->size, flt,ne_tensor); break;
			case eq_scalar: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i],A->size, flt,eq_scalar); break;
			case lt_scalar: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i],A->size, flt,lt_scalar); break;
			case gt_scalar: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i],A->size, flt,gt_scalar); break;
			case ge_scalar: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i],A->size, flt,ge_scalar); break;
			case le_scalar: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i],A->size, flt,le_scalar); break;
			case ne_scalar: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i],A->size, flt,ne_scalar); break;
			case dropout_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i],A->size, flt,dropout_tensor); break;
			case eq_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->batches, A->rows, A->rows*A->cols, eq_vec); break;
			case lt_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->batches, A->rows, A->rows*A->cols, lt_vec); break;
			case gt_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->batches, A->rows, A->rows*A->cols, gt_vec); break;
			case le_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->batches, A->rows, A->rows*A->cols, le_vec); break;
			case ge_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->batches, A->rows, A->rows*A->cols, ge_vec); break;
			case ne_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->batches, A->rows, A->rows*A->cols, ne_vec); break;
			case add_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->batches, A->rows, A->rows*A->cols, add_vec); break;
			case sub_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->batches, A->rows, A->rows*A->cols, sub_vec); break;
			case mul_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->batches, A->rows, A->rows*A->cols, mul_vec); break;
			case div_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->batches, A->rows, A->rows*A->cols, div_vec); break;

			default: throw "Unsupported operation!";
		}
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}


void synchronize(Tensor *A, Tensor *out, int myid, int copyid, hipStream_t stream,Operation_t ops)
{
	int block_size = (A->rows*A->cols/THREADS_PER_BLOCKS) + 1;
	CUDA_CHECK_RETURN(hipSetDevice(myid));
	kElementWise<<<block_size,THREADS_PER_BLOCKS,0,stream>>>(A->data_gpus[myid],A->data_gpus[copyid],out->data_gpus[myid],A->size,0.0f,ops);
	CUDA_CHECK_RETURN(hipPeekAtLastError());
}


