#include "hip/hip_runtime.h"
#include <basics.cuh>
#include <assert.h>

using std::cout;
using std::endl;

Slice *emptySlice()
{
	Slice *out = new Slice();
	out->batch_start = 0;
	out->batch_stop = INT_MAX;
	out->map_start = 0;
	out->map_stop = INT_MAX;
	out->row_start = 0;
	out->row_stop = INT_MAX;
	out->col_start = 0;
	out->col_stop = INT_MAX;

	return out;
}



int *get_split_shape(int batches, int maps, int rows, int cols,int split_axis,int gpuidx)
{
	int *ret = new int[4];
	ret[0] = batches; ret[1] = maps; ret[2] = rows; ret[3] = cols;
	if(split_axis==-1){ return ret; }

	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	int size = ret[split_axis];
	int split_size = 1+ (size/gpus);
	assert(split_size >= gpus);
	int split_offsize = size - ((gpus-1)*split_size);
	if(size % gpus == 0)
	{
		split_size -=1;
		split_offsize =split_size;
	}

	if(size == gpus){split_offsize = 1; split_size = 1;}
	if(gpuidx==gpus-1){ret[split_axis] = split_offsize; }
	else{ret[split_axis] = split_size;}

	return ret;

}

//go around export "C" with this declaration
template <typename T>
TensorTemplate<T>* empty_template(int batches, int maps, int rows, int cols, int split_axis)
{

	TensorTemplate<T> *out = new TensorTemplate<T>();
	int size = batches*maps*rows*cols;
	size_t bytes = size*sizeof(T);
	out->batches = batches;
	out->maps = maps;
	out->rows = rows;
	out->cols = cols;
	out->bytes = bytes;
	out->size = size;
	out->isCUDA = 1;
	out->splitAxis = split_axis;

	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		int *shape = get_split_shape(out->batches,out->maps, out->rows,out->cols, split_axis, i);

		out->shape_gpus.push_back(shape);
		out->size_gpus.push_back(shape[0]*shape[1]*shape[2]*shape[3]);
		out->bytes_gpus.push_back(shape[0]*shape[1]*shape[2]*shape[3]*sizeof(T));


		T *gpu_data;
		CUDA_CHECK_RETURN(hipMalloc((void**)&gpu_data, out->bytes_gpus.back()));

		if(i == 0){ out->data = gpu_data; }
		out->data_gpus.push_back(gpu_data);
	}

	CUDA_CHECK_RETURN(hipSetDevice(0));

	return out;
}

Tensor *empty_like(Tensor *A){ return empty(A->batches, A->maps, A->rows, A->cols, A->splitAxis); }
Tensor *empty(int batches, int maps, int rows, int cols){ return empty(batches, maps, rows, cols, -1); }
Tensor *empty(int batches, int maps, int rows, int cols, int split_axis)
{ return (Tensor*)empty_template<float>(batches, maps, rows, cols, split_axis); }

CharTensor *empty_char_like(Tensor *A){ return empty_char(A->batches, A->maps, A->rows, A->cols, A->splitAxis); }
CharTensor *empty_char(int batches, int maps, int rows, int cols){ return empty_char(batches, maps, rows, cols, -1); }
CharTensor *empty_char(int batches, int maps, int rows, int cols, int split_axis)
{ return (CharTensor*)empty_template<unsigned char>(batches, maps, rows, cols, split_axis); }


UIntTensor *empty_uint_like(Tensor *A){ return empty_uint(A->batches, A->maps, A->rows, A->cols/32, A->splitAxis); }
UIntTensor *empty_uint(int batches, int maps, int rows, int cols){ return empty_uint(batches, maps, rows, cols, -1); }
UIntTensor *empty_uint(int batches, int maps, int rows, int cols, int split_axis)
{ return (UIntTensor*)empty_template<unsigned int>(batches, maps, rows, cols, split_axis); }


void slice_axis(Tensor *A, Tensor *out)
{
	//only row slice supported right now
	assert(out->splitAxis == 2 && A->splitAxis == -1);
	Slice *S = emptySlice();
	S->batch_stop = A->batches;
	S->map_stop = A->maps;
	S->col_stop = A->cols;
	S->row_stop = 0;

	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		S->row_stop += out->shape_gpus[i][2];
		CUDA_CHECK_RETURN(hipSetDevice(i));
		//this is a complete mess, an evil monster, but will do for now
		kSlice<<<dim3(A->shape_gpus[i][0], A->shape_gpus[i][1],1),dim3(32,32,1)>>>(A->data_gpus[i],out->data_gpus[i],
				S->batch_start, S->batch_stop,
				S->map_start, S->map_stop,
				S->row_start, S->row_stop,
				S->col_start, S->col_stop,
				A->shape_gpus[i][2],A->shape_gpus[i][3],
				out->shape_gpus[i][0],out->shape_gpus[i][1],
				out->shape_gpus[i][3],out->shape_gpus[i][2], 1);

		CUDA_CHECK_RETURN(hipPeekAtLastError());

		S->row_start += out->shape_gpus[i][2];
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}

void stack_axis(Tensor *A, Tensor *out)
{
	//only row slice supported right now
		assert(out->splitAxis == -1 && A->splitAxis == 2);
		Slice *S = emptySlice();
		S->batch_stop = A->batches;
		S->map_stop = A->maps;
		S->col_stop = A->cols;
		S->row_stop = 0;

		int gpus = 0;
		CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
		for(int i = 0; i < gpus; i++)
		{
			S->row_stop = 0;
			S->row_start = 0;
			for(int j = 0; j < gpus; j++)
			{
				S->row_stop += A->shape_gpus[j][2];
				CUDA_CHECK_RETURN(hipSetDevice(i));
				//this is a complete mess, an evil monster, but will do for now
					kSlice<<<dim3(out->shape_gpus[i][0], out->shape_gpus[i][1],1),dim3(32,32,1)>>>(A->data_gpus[j],out->data_gpus[i],
								S->batch_start, S->batch_stop,
								S->map_start, S->map_stop,
								S->row_start, S->row_stop,
								S->col_start, S->col_stop,
								out->shape_gpus[i][2],out->shape_gpus[i][3],
								A->shape_gpus[j][0],A->shape_gpus[j][1],
								A->shape_gpus[j][3],A->shape_gpus[j][2], 0);
				CUDA_CHECK_RETURN(hipPeekAtLastError());

				S->row_start += A->shape_gpus[j][2];
			}
		}
		CUDA_CHECK_RETURN(hipSetDevice(0));
}

float *empty_pinned(int batches, int maps, int rows, int cols, float *cpu_buffer)
{
	Tensor *out = new Tensor();
	int size = batches*maps*rows*cols;
	float *pinned_data;
	size_t bytes = size*sizeof(float);
	CUDA_CHECK_RETURN(hipHostAlloc(&pinned_data, bytes, hipHostMallocPortable));
	if(cpu_buffer)
		CUDA_CHECK_RETURN(hipMemcpy(pinned_data,cpu_buffer,bytes,hipMemcpyDefault));
	out->batches = batches;
	out->maps = maps;
	out->rows = rows;
	out->cols = cols;
	out->bytes = bytes;
	out->size = size;
	out->isCUDA = 1;
	out->splitAxis = -1;
	out->data = pinned_data;

	return pinned_data;
}

Tensor *zeros(int batches, int maps, int rows, int cols){ return zeros(batches, maps, rows, cols, -1); }
Tensor *zeros(int batches, int maps, int rows, int cols, int split_axis)
{
	Tensor *out = empty(batches,maps,rows,cols,split_axis);
	elementWise(out, NULL,NULL,0.0f,fill);
	return out;
}

Tensor *ones(int batches, int maps, int rows, int cols)
{
	Tensor *out = empty(batches,maps,rows,cols);
	elementWise(out, NULL,NULL,1.0f,fill);
	return out;
}


Tensor *T(Tensor *A)
{
	Tensor *out = empty(A->batches,A->maps,A->cols,A->rows);
	T(A,out, 2,3);
	out->rows = A->cols;
	out->cols = A->rows;
	return out;
}


void T(Tensor *A, Tensor *out,  int rows_idx, int cols_idx)
{
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		int rows = A->shape_gpus[i][rows_idx];
		int cols= A->shape_gpus[i][cols_idx];

		// setup execution parameters
		int grid_x = rows / COPY_BLOCK_SIZE;
		if (rows  % COPY_BLOCK_SIZE)
			grid_x++;

		int grid_y = cols / COPY_BLOCK_SIZE;
		if (cols % COPY_BLOCK_SIZE)
			grid_y++;

		dim3 grid(grid_x, grid_y, A->maps);
		dim3 threads(COPY_BLOCK_SIZE, COPY_BLOCK_SIZE, 1);
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kTransposeTensor<<< grid, threads >>>(A->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][0], rows, cols);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}

Tensor *to_col_major(Tensor *A)
{
  Tensor *out = empty_like(A);
  T(A, out, 3,2);

  return out;
}

void to_col_major(Tensor *A, Tensor *out)
{
	T(A, out, 3,2);
}

Tensor *to_row_major(Tensor *A)
{
	Tensor *out = empty_like(A);
	T(A, out, 2,3);

  return out;
}

Tensor *tocpu(Tensor *A, float *cpu_buffer)
{
	Tensor *temp = to_row_major(A);
	Tensor *out = new Tensor();

	CUDA_CHECK_RETURN(hipMemcpy(cpu_buffer,temp->data_gpus[0],temp->bytes_gpus[0],hipMemcpyDefault));
	out->batches = temp->batches;
	out->maps = temp->maps;
	out->rows = temp->rows;
	out->cols = temp->cols;
	out->bytes = temp->bytes;
	out->size = temp->size;
	out->data = cpu_buffer;
	out->isCUDA = 0;
	out->splitAxis = -1;

	CUDA_CHECK_RETURN(hipFree(temp->data));
	delete temp;


	return out;
}


void print_slice(Slice *S)
{
	cout << "batch: " << S->batch_start << " to " << S->batch_stop << endl;
	cout << "map: " << S->map_start << " to " << S->map_stop << endl;
	cout << "row: " << S->row_start << " to " << S->row_stop << endl;
	cout << "col: " << S->col_start << " to " << S->col_stop << endl;
}

void print_shape(int *shape)
{
	cout << shape[0] << "x" << shape[1] << "x" << shape[2]<< "x" << shape[3] << endl;
}

float print_free_memory()
{
	size_t total, free;
	hipMemGetInfo(&free,&total);

	cout << "Free GB: " << ((float)free)/1024./1024./1024. << endl;

	return ((float)free)/1024./1024./1024.;
}

void print_tensor_shape(Tensor *A)
{
	for(int i = 0; i < A->data_gpus.size(); i++)
		print_shape(A->shape_gpus[i]);
}


void togpu(Tensor *out, float *cpu_buffer){ togpu(out, cpu_buffer, -1); }
void togpu(Tensor *out, float *cpu_buffer, int split_axis)
{
	Tensor *temp = empty(out->batches,out->maps,out->rows,out->cols);
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));

	if(split_axis==2)
	{
		for(int i = 0; i < gpus; i++){ CUDA_CHECK_RETURN(hipMemcpy(temp->data_gpus[i],cpu_buffer,temp->bytes_gpus[i],hipMemcpyDefault)); }
		Tensor *temp2 = to_col_major(temp);
		Slice *S = emptySlice();
		S->batch_stop = temp->batches;
		S->map_stop = temp->maps;
		S->col_stop = temp->cols;
		S->row_stop = 0;
		for(int i = 0; i < gpus; i++)
		{

			S->row_stop += out->shape_gpus[i][2];
			CUDA_CHECK_RETURN(hipSetDevice(i));
			//print_shape(temp2->shape_gpus[i]);
			//cout << temp2->size_gpus[i] << endl;
			//cout << temp2->bytes_gpus[i] << endl;
			kSlice<<<dim3(temp2->shape_gpus[i][0], temp2->shape_gpus[i][1],1),dim3(32,32,1)>>>(temp2->data_gpus[i],out->data_gpus[i],
					S->batch_start, S->batch_stop,
					S->map_start, S->map_stop,
					S->row_start, S->row_stop,
					S->col_start, S->col_stop,
					temp2->shape_gpus[i][2],temp2->shape_gpus[i][3],
					out->shape_gpus[i][0],out->shape_gpus[i][1],
					out->shape_gpus[i][3],out->shape_gpus[i][2], 1);
			CUDA_CHECK_RETURN(hipPeekAtLastError());



			S->row_start += out->shape_gpus[i][2];
		}
		temp2->freeTensor();
	}
	else if (split_axis == -1)
	{
		for(int i = 0; i < gpus; i++){ CUDA_CHECK_RETURN(hipMemcpy(out->data_gpus[i],cpu_buffer,out->bytes_gpus[i],hipMemcpyDefault)); }
		to_col_major(out, temp);
		for(int i = 0; i < gpus; i++){ CUDA_CHECK_RETURN(hipMemcpy(out->data_gpus[i],temp->data_gpus[i],out->bytes_gpus[i],hipMemcpyDefault)); }
	}
	else
	{
		throw "uden!";
	}

	CUDA_CHECK_RETURN(hipSetDevice(0));
	temp->freeTensor();
}


Tensor *applySliceFunc(Tensor *A, Slice *S)
{
	Tensor *out = zeros(S->batch_stop-S->batch_start,
						S->map_stop-S->map_start,
						S->row_stop-S->row_start,
						S->col_stop-S->col_start);

	applySliceFunc(A, S, out);

	return out;

}

void applySliceFunc(Tensor *A, Slice *S, Tensor *out)
{
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));

	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kSlice<<<dim3(A->shape_gpus[i][0], A->shape_gpus[i][1],1),dim3(32,32,1)>>>(A->data_gpus[i],out->data_gpus[i],
				S->batch_start, S->batch_stop,
				S->map_start, S->map_stop,
				S->row_start, S->row_stop,
				S->col_start, S->col_stop,
				A->shape_gpus[i][2],A->shape_gpus[i][3],
				out->shape_gpus[i][0],out->shape_gpus[i][1],
				out->shape_gpus[i][3],out->shape_gpus[i][2],1);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}

	CUDA_CHECK_RETURN(hipSetDevice(0));


}



Tensor *elementWise(Tensor *A, Tensor *B, Operation_t ops){ return elementWise(A, B, 0.0f, ops); }
Tensor *elementWise(Tensor *A, Tensor *B, float flt, Operation_t ops)
{ Tensor *out = empty_like(A); elementWise(A, B, out, flt, ops); return out; }
void elementWise(Tensor *A, Tensor *B, Tensor *out, Operation_t ops){ elementWise(A,B,out, 0.0f, ops); }
void elementWise(Tensor *A, Tensor *B, Tensor *out, float flt, Operation_t ops)
{
		int gpus = 0;
		CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
		for(int i = 0; i < gpus; i++)
		{
			int block_size = (A->shape_gpus[i][2]*A->shape_gpus[i][3]/THREADS_PER_BLOCKS) + 1;
			CUDA_CHECK_RETURN(hipSetDevice(i));
			if(B && out)
				kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i],A->size_gpus[i], flt, ops);
			else if(out)
				kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i],A->size_gpus[i], flt, ops);
			else
				kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, NULL,A->size_gpus[i], flt, ops);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			if(ops == print){ CUDA_CHECK_RETURN(hipDeviceSynchronize());}
		}
		CUDA_CHECK_RETURN(hipSetDevice(0));

}
Tensor *vectorWise(Tensor *A, Tensor *B, Operation_t ops){ return vectorWise(A,B,0.0f,ops); }
Tensor *vectorWise(Tensor *A, Tensor *B, float flt, Operation_t ops)
{ Tensor *out = empty_like(A); vectorWise(A, B, out, flt, ops);	return out; }
void vectorWise(Tensor *A, Tensor *B, Tensor *out, Operation_t ops){ vectorWise(A,B,out,0.0f,ops); }
void vectorWise(Tensor *A, Tensor *B, Tensor *out, float flt, Operation_t ops)
{
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		int block_size = (A->shape_gpus[i][2]*A->shape_gpus[i][3]/THREADS_PER_BLOCKS) + 1;
		dim3 grid(block_size, A->shape_gpus[i][1],A->shape_gpus[i][0]);
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][0], A->shape_gpus[i][2], A->shape_gpus[i][3]*A->shape_gpus[i][2], ops);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}


Tensor *softmax(Tensor *A){ Tensor *out = empty_like(A); softmax(A,out); return out; }
void softmax(Tensor *A, Tensor *out)
{
	dim3 grids(A->batches, A->maps);
	dim3 threads(A->rows > THREADS_PER_BLOCKS ? THREADS_PER_BLOCKS : A->rows, 1);
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kSoftMax<<<grids,threads >>>(A->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][2], A->shape_gpus[i][3]);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}

Tensor *argmax(Tensor *A){ Tensor *out = empty(A->batches,A->maps,A->rows,1, A->splitAxis); argmax(A,out); return out; }
void argmax(Tensor *A, Tensor *out)
{
	dim3 grids(A->batches, A->maps);
	dim3 threads(A->rows > THREADS_PER_BLOCKS ? THREADS_PER_BLOCKS : A->rows, 1);
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kArgmax<<<grids,threads >>>(A->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][2], A->shape_gpus[i][3]);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}

void weightUpdate(Tensor *RMS, Tensor *grad, float RMS_multiplier, float learning_rate, int batch_size, weightUpdate_t strategy)
{

	int blocks = (RMS->size/THREADS_PER_BLOCKS) + 1;
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kWeightUpdate<<<blocks,THREADS_PER_BLOCKS>>>(RMS->data_gpus[i], grad->data_gpus[i], RMS_multiplier, learning_rate, batch_size, RMS->size, strategy);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}




float thrust_reduce(Tensor *A, Operation_t strategy)
{
	float value = 0;
	if(A->splitAxis == -1)
	{
		thrust::device_ptr<float> ptr(A->data);
		switch(strategy)
		{
			case sum_tensor: value = thrust::reduce(ptr, ptr+A->size); break;
			case max_tensor: value = thrust::reduce(ptr, ptr+A->size,-1.0f, thrust::maximum<float>()); break;
			case min_tensor: value = thrust::reduce(ptr, ptr+A->size,-1.0f, thrust::minimum<float>()); break;
		}

	}
	else
	{
		switch(strategy){ case max_tensor: value = -FLT_MAX; break; case min_tensor: value = FLT_MAX; break; }
		int gpus = 0;
		CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
		for(int i = 0; i < gpus; i++)
		{
			CUDA_CHECK_RETURN(hipSetDevice(i));
			thrust::device_ptr<float> ptr(A->data_gpus[i]);
			switch(strategy)
			{
				case sum_tensor: value += thrust::reduce(ptr, ptr+A->size_gpus[i]); break;
				case max_tensor: value = fmax(value,thrust::reduce(ptr, ptr+A->size_gpus[i],-1.0f, thrust::maximum<float>())); break;
				case min_tensor: value = fmin(value,thrust::reduce(ptr, ptr+A->size_gpus[i],-1.0f, thrust::minimum<float>())); break;
			}
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		}
		CUDA_CHECK_RETURN(hipSetDevice(0));
	}

	return value;
}

void compression_8bit(Tensor *tbl_flt, Tensor *A, float precision,  CharTensor *out)
{
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		int blocks = (A->size_gpus[i]/THREADS_PER_BLOCKS) + 1;
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kCompression_8bit<<<blocks,THREADS_PER_BLOCKS>>>(tbl_flt->data_gpus[i], A->data_gpus[i], precision, A->size_gpus[i], out->data_gpus[i]);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));

}



void decompression_8bit(Tensor *tbl_flt, CharTensor *A, float precision,  Tensor *out)
{
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		int blocks = (A->size_gpus[i]/THREADS_PER_BLOCKS) + 1;
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kDecompression_8bit<<<blocks,THREADS_PER_BLOCKS>>>(tbl_flt->data_gpus[i],  A->data_gpus[i], precision, A->size_gpus[i], out->data_gpus[i]);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}


void compression_1bit(Tensor *A_with_errors, Tensor *errors, Tensor *avgPos, Tensor *avgNeg, UIntTensor *out)
{
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		int blocks = (A_with_errors->size_gpus[i]/THREADS_PER_BLOCKS) + 1;
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kCompression_1bit<<<blocks,THREADS_PER_BLOCKS>>>(A_with_errors->data_gpus[i], errors->data_gpus[i],avgPos->data_gpus[i], avgNeg->data_gpus[i], out->data_gpus[i],A_with_errors->shape_gpus[i][2],A_with_errors->shape_gpus[i][3]);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));

}



void decompression_1bit(UIntTensor *quant, Tensor *errors, Tensor *avgPos, Tensor *avgNeg, Tensor *out)
{
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		int blocks = (out->size_gpus[i]/THREADS_PER_BLOCKS) + 1;
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kDecompression_1bit<<<blocks,THREADS_PER_BLOCKS>>>(quant->data_gpus[i], errors->data_gpus[i],avgPos->data_gpus[i], avgNeg->data_gpus[i], out->data_gpus[i],out->shape_gpus[i][2],out->shape_gpus[i][3]);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}





void reduceRow(Tensor *A, Tensor *out, Operation_t ops)
{
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		int blocks = max(256,(A->shape_gpus[i][2]));
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kReduceRow<<<blocks,256>>>(A->data_gpus[i], out->data_gpus[i], A->rows, A->cols);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));

}

hipEvent_t* tick()
{
    hipEvent_t* startstop;
    startstop = (hipEvent_t*)malloc(2*sizeof(hipEvent_t));
    hipEventCreate(&startstop[0]);
    hipEventCreate(&startstop[1]);
    hipEventRecord(startstop[0], 0);

    return startstop;
}
float tock(hipEvent_t* startstop){ return tock(startstop, "Time for the kernel(s): "); }
float tock(hipEvent_t* startstop, std::string text)
{
	float time;
	hipEventRecord(startstop[1], 0);
	hipEventSynchronize(startstop[1]);
	hipEventElapsedTime(&time, startstop[0], startstop[1]);
	printf((text + ": %f ms.\n").c_str(), time);
	return time;
}
float tock(std::string text, float tocks)
{
	printf((text + ": %f ms.\n").c_str(), tocks);
	return tocks;
}
float tock(hipEvent_t* startstop, float tocks)
{
	float time;
	hipEventRecord(startstop[1], 0);
	hipEventSynchronize(startstop[1]);
	hipEventElapsedTime(&time, startstop[0], startstop[1]);

	return time+tocks;
}

