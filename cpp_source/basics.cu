#include "hip/hip_runtime.h"
#include <basics.cuh>
#include <assert.h>

using std::cout;
using std::endl;

Slice *emptySlice()
{
	Slice *out = new Slice();
	out->batch_start = 0;
	out->batch_stop = INT_MAX;
	out->map_start = 0;
	out->map_stop = INT_MAX;
	out->row_start = 0;
	out->row_stop = INT_MAX;
	out->col_start = 0;
	out->col_stop = INT_MAX;

	return out;
}

Tensor *empty_like(Tensor *A)
{
	return empty(A->batches, A->maps, A->rows, A->cols, A->splitAxis);
}

int *get_split_shape(int batches, int maps, int rows, int cols,int split_axis,int gpuidx)
{
	int *ret = new int[4];
	ret[0] = batches; ret[1] = maps; ret[2] = rows; ret[3] = cols;
	if(split_axis==-1){ return ret; }

	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	int size = ret[split_axis];
	int split_size = 1+ (size/gpus);
	assert(split_size >= gpus);
	int split_offsize = size - ((gpus-1)*split_size);
	if(size % gpus == 0)
	{
		split_size -=1;
		split_offsize =split_size;
	}

	if(size == gpus){split_offsize = 1; split_size = 1;}
	if(gpuidx==gpus-1){ret[split_axis] = split_offsize; }
	else{ret[split_axis] = split_size;}

	return ret;

}


Tensor *empty(int batches, int maps, int rows, int cols){ return empty(batches, maps, rows, cols, -1); }
Tensor *empty(int batches, int maps, int rows, int cols, int split_axis)
{

	Tensor *out = new Tensor();
	int size = batches*maps*rows*cols;
	size_t bytes = size*sizeof(float);
	out->batches = batches;
	out->maps = maps;
	out->rows = rows;
	out->cols = cols;
	out->bytes = bytes;
	out->size = size;
	out->isCUDA = 1;
	out->splitAxis = split_axis;

	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		int *shape = get_split_shape(out->batches,out->maps, out->rows,out->cols, split_axis, i);

		out->shape_gpus.push_back(shape);
		out->size_gpus.push_back(shape[0]*shape[1]*shape[2]*shape[3]);
		out->bytes_gpus.push_back(shape[0]*shape[1]*shape[2]*shape[3]*sizeof(float));


		float *gpu_data;
		CUDA_CHECK_RETURN(hipMalloc((void**)&gpu_data, out->bytes_gpus.back()));

		if(i == 0){ out->data = gpu_data; }
		out->data_gpus.push_back(gpu_data);
	}

	CUDA_CHECK_RETURN(hipSetDevice(0));

	return out;
}



void slice_or_stack_axis(Tensor *A, Tensor *out)
{
	//only row slice supported right now
	assert((out->splitAxis == -1 && A->splitAxis == 2) ||
			(out->splitAxis == 2 && A->splitAxis == -1));
	int forward_split = out->splitAxis == 2;
	Slice *S = emptySlice();
	S->batch_stop = A->batches;
	S->map_stop = A->maps;
	S->col_stop = A->cols;
	S->row_stop = 0;

	int gpus = 0;
	int idx = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		if(forward_split == 0){ S->row_stop = 0; S->row_start = 0;}
		for(int j = 0; j < (forward_split == 1 ? 1 : gpus); j++)
		{
			if(forward_split == 0) idx = j;
			else idx = i;
			S->row_stop += out->splitAxis == 2 ? out->shape_gpus[i][2] : A->shape_gpus[idx][2];
			CUDA_CHECK_RETURN(hipSetDevice(i));
			//this is a complete mess, an evil monster, but will do for now
			if(forward_split == 1)
			kSlice<<<dim3(A->shape_gpus[idx][0], A->shape_gpus[i][1],1),dim3(32,32,1)>>>(A->data_gpus[idx],out->data_gpus[i],
					S->batch_start, S->batch_stop,
					S->map_start, S->map_stop,
					S->row_start, S->row_stop,
					S->col_start, S->col_stop,
					A->shape_gpus[i][2],A->shape_gpus[i][3],
					out->shape_gpus[i][0],out->shape_gpus[i][1],
					out->shape_gpus[i][3],out->shape_gpus[i][2], forward_split);
			else
				kSlice<<<dim3(out->shape_gpus[i][0], out->shape_gpus[i][1],1),dim3(32,32,1)>>>(A->data_gpus[idx],out->data_gpus[i],
							S->batch_start, S->batch_stop,
							S->map_start, S->map_stop,
							S->row_start, S->row_stop,
							S->col_start, S->col_stop,
							out->shape_gpus[i][2],out->shape_gpus[i][3],
							A->shape_gpus[idx][0],A->shape_gpus[idx][1],
							A->shape_gpus[idx][3],A->shape_gpus[idx][2], forward_split);
			CUDA_CHECK_RETURN(hipPeekAtLastError());

			S->row_start += out->splitAxis == 2 ? out->shape_gpus[i][2] : A->shape_gpus[idx][2];
		}
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}

void stack_axis(Tensor *A, Tensor *out)
{
	//only row slice supported right now
	assert((out->splitAxis == -1 && A->splitAxis == 2));
	Slice *S = emptySlice();
	S->batch_stop = out->batches;
	S->map_stop = out->maps;
	S->col_stop = out->cols;
	S->row_stop = 0;

	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		 S->row_stop = 0;
		 S->row_start = 0;
		for(int j = 0; j < gpus; j++)
		{
			S->row_stop += A->shape_gpus[j][2];
			CUDA_CHECK_RETURN(hipSetDevice(i));
			kSlice<<<dim3(A->shape_gpus[j][0], A->shape_gpus[j][1],1),dim3(32,32,1)>>>(A->data_gpus[j],out->data_gpus[i],
					S->batch_start, S->batch_stop,
					S->map_start, S->map_stop,
					S->row_start, S->row_stop,
					S->col_start, S->col_stop,
					A->shape_gpus[j][2],A->shape_gpus[j][3],
					out->shape_gpus[i][0],out->shape_gpus[i][1],
					out->shape_gpus[i][3],out->shape_gpus[i][2], 0);
			CUDA_CHECK_RETURN(hipPeekAtLastError());

			S->row_start += out->splitAxis == 2 ? out->shape_gpus[i][2] : A->shape_gpus[i][2];
		}
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}

float *empty_pinned(int batches, int maps, int rows, int cols, float *cpu_buffer)
{
	Tensor *out = new Tensor();
	int size = batches*maps*rows*cols;
	float *pinned_data;
	size_t bytes = size*sizeof(float);
	CUDA_CHECK_RETURN(hipHostAlloc(&pinned_data, bytes, hipHostMallocPortable));
	if(cpu_buffer)
		CUDA_CHECK_RETURN(hipMemcpy(pinned_data,cpu_buffer,bytes,hipMemcpyDefault));
	out->batches = batches;
	out->maps = maps;
	out->rows = rows;
	out->cols = cols;
	out->bytes = bytes;
	out->size = size;
	out->isCUDA = 1;
	out->splitAxis = -1;
	out->data = pinned_data;

	return pinned_data;
}

Tensor *zeros(int batches, int maps, int rows, int cols){ return zeros(batches, maps, rows, cols, -1); }
Tensor *zeros(int batches, int maps, int rows, int cols, int split_axis)
{
	Tensor *out = empty(batches,maps,rows,cols,split_axis);
	return fill_with_number(out, 0.0f);
}

Tensor *ones(int batches, int maps, int rows, int cols)
{
	Tensor *out = empty(batches,maps,rows,cols);
	return fill_with_number(out, 1.0f);
}

Tensor *fill_with_number(Tensor *A, float fill_value)
{
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		thrust::device_ptr<float> ptr_dev(A->data_gpus[i]);
		thrust::fill(ptr_dev, ptr_dev + A->size_gpus[i],fill_value);
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));

	return A;
}



Tensor *T(Tensor *A)
{
	Tensor *out = empty(A->batches,A->maps,A->cols,A->rows);
	T(A,out, A->rows,A->cols);
	out->rows = A->cols;
	out->cols = A->rows;
	return out;
}


void T(Tensor *A, Tensor *out,  int rows, int cols)
{
	// setup execution parameters
	int grid_x = rows / COPY_BLOCK_SIZE;
	if (rows  % COPY_BLOCK_SIZE)
		grid_x++;

	int grid_y = cols / COPY_BLOCK_SIZE;
	if (cols % COPY_BLOCK_SIZE)
		grid_y++;

	dim3 grid(grid_x, grid_y, A->maps);
	dim3 threads(COPY_BLOCK_SIZE, COPY_BLOCK_SIZE, 1);
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kTransposeTensor<<< grid, threads >>>(A->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][0], rows, cols);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}

Tensor *to_col_major(Tensor *A)
{
  Tensor *out = empty_like(A);
  T(A, out, A->cols,A->rows);

  return out;
}

void to_col_major(Tensor *A, Tensor *out)
{
	T(A, out, A->cols,A->rows);
}

Tensor *to_row_major(Tensor *A)
{
	Tensor *out = empty_like(A);
	T(A, out, A->rows,A->cols);

  return out;
}



Tensor *tocpu(Tensor *A, float *cpu_buffer)
{
	Tensor *temp = to_row_major(A);
	Tensor *out = new Tensor();

	CUDA_CHECK_RETURN(hipMemcpy(cpu_buffer,temp->data_gpus[0],temp->bytes_gpus[0],hipMemcpyDefault));
	out->batches = temp->batches;
	out->maps = temp->maps;
	out->rows = temp->rows;
	out->cols = temp->cols;
	out->bytes = temp->bytes;
	out->size = temp->size;
	out->data = cpu_buffer;
	out->isCUDA = 0;
	out->splitAxis = -1;

	CUDA_CHECK_RETURN(hipFree(temp->data));
	delete temp;


	return out;
}


void print_slice(Slice *S)
{
	cout << "batch: " << S->batch_start << " to " << S->batch_stop << endl;
	cout << "map: " << S->map_start << " to " << S->map_stop << endl;
	cout << "row: " << S->row_start << " to " << S->row_stop << endl;
	cout << "col: " << S->col_start << " to " << S->col_stop << endl;
}

void print_shape(int *shape)
{
	cout << shape[0] << "x" << shape[1] << "x" << shape[2]<< "x" << shape[3] << endl;
}

void print_free_memory()
{
	size_t total, free;
	hipMemGetInfo(&free,&total);
	cout << "Free GB: " << ((double)free)/1024./1024./1024. << endl;
}

void print_tensor_shape(Tensor *A)
{
	for(int i = 0; i < A->data_gpus.size(); i++)
		print_shape(A->shape_gpus[i]);
}


void togpu(Tensor *out, float *cpu_buffer){ togpu(out, cpu_buffer, -1); }
void togpu(Tensor *out, float *cpu_buffer, int split_axis)
{
	Tensor *temp = empty(out->batches,out->maps,out->rows,out->cols);
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));

	if(split_axis==2)
	{
		for(int i = 0; i < gpus; i++){ CUDA_CHECK_RETURN(hipMemcpy(temp->data_gpus[i],cpu_buffer,temp->bytes_gpus[i],hipMemcpyDefault)); }
		Tensor *temp2 = to_col_major(temp);
		Slice *S = emptySlice();
		S->batch_stop = temp->batches;
		S->map_stop = temp->maps;
		S->col_stop = temp->cols;
		S->row_stop = 0;
		for(int i = 0; i < gpus; i++)
		{

			S->row_stop += out->shape_gpus[i][2];
			CUDA_CHECK_RETURN(hipSetDevice(i));
			//print_shape(temp2->shape_gpus[i]);
			//cout << temp2->size_gpus[i] << endl;
			//cout << temp2->bytes_gpus[i] << endl;
			kSlice<<<dim3(temp2->shape_gpus[i][0], temp2->shape_gpus[i][1],1),dim3(32,32,1)>>>(temp2->data_gpus[i],out->data_gpus[i],
					S->batch_start, S->batch_stop,
					S->map_start, S->map_stop,
					S->row_start, S->row_stop,
					S->col_start, S->col_stop,
					temp2->shape_gpus[i][2],temp2->shape_gpus[i][3],
					out->shape_gpus[i][0],out->shape_gpus[i][1],
					out->shape_gpus[i][3],out->shape_gpus[i][2], 1);
			CUDA_CHECK_RETURN(hipPeekAtLastError());



			S->row_start += out->shape_gpus[i][2];
		}
		temp2->freeTensor();
	}
	else if (split_axis == -1)
	{
		for(int i = 0; i < gpus; i++){ CUDA_CHECK_RETURN(hipMemcpy(out->data_gpus[i],cpu_buffer,out->bytes_gpus[i],hipMemcpyDefault)); }
		to_col_major(out, temp);
		for(int i = 0; i < gpus; i++){ CUDA_CHECK_RETURN(hipMemcpy(out->data_gpus[i],temp->data_gpus[i],out->bytes_gpus[i],hipMemcpyDefault)); }
	}
	else
	{
		throw 'uden!';
	}

	CUDA_CHECK_RETURN(hipSetDevice(0));
	temp->freeTensor();
}


Tensor *applySliceFunc(Tensor *A, Slice *S)
{
	Tensor *out = zeros(S->batch_stop-S->batch_start,
						S->map_stop-S->map_start,
						S->row_stop-S->row_start,
						S->col_stop-S->col_start);

	applySliceFunc(A, S, out);

	return out;

}

void applySliceFunc(Tensor *A, Slice *S, Tensor *out)
{
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kSlice<<<dim3(A->shape_gpus[i][0], A->shape_gpus[i][1],1),dim3(32,32,1)>>>(A->data_gpus[i],out->data_gpus[i],
				S->batch_start, S->batch_stop,
				S->map_start, S->map_stop,
				S->row_start, S->row_stop,
				S->col_start, S->col_stop,
				A->shape_gpus[i][2],A->shape_gpus[i][3],
				out->shape_gpus[i][0],out->shape_gpus[i][1],
				out->shape_gpus[i][3],out->shape_gpus[i][2],1);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}

	CUDA_CHECK_RETURN(hipSetDevice(0));


}

Tensor *applyFunc(Tensor *A, Tensor *B, Operation_t ops){ return applyFunc(A,B,0.0f,ops); }
Tensor *applyFunc(Tensor *A, Tensor *B, float flt, Operation_t ops)
{
	Tensor *out = empty_like(A);
	applyFunc(A, B, out, flt, ops);

	return out;
}

void applyFunc(Tensor *A, Tensor *B, Tensor *out, Operation_t ops){ applyFunc(A,B,out,0.0f,ops); }
void applyFunc(Tensor *A, Tensor *B, Tensor *out, float flt, Operation_t ops)
{
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		int block_size = (A->shape_gpus[i][2]*A->shape_gpus[i][3]/THREADS_PER_BLOCKS) + 1;
		dim3 grid(block_size, A->shape_gpus[i][1],A->shape_gpus[i][0]);
		CUDA_CHECK_RETURN(hipSetDevice(i));
		switch(ops)
		{
			case copy: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size_gpus[i], flt, copy); break;
			case add_scalar: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size_gpus[i], flt, add_scalar); break;
			case mul_scalar: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size_gpus[i], flt, mul_scalar); break;
			case add_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->size_gpus[i], flt, add_tensor); break;
			case sub_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->size_gpus[i], flt, sub_tensor); break;
			case mul_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->size_gpus[i], flt, mul_tensor); break;
			case div_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->size_gpus[i], flt, div_tensor); break;
			case abs_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size_gpus[i], flt, abs_tensor); break;
			case log_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size_gpus[i], flt, log_tensor); break;
			case exp_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size_gpus[i], flt, exp_tensor); break;
			case pow_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size_gpus[i], flt, pow_tensor); break;
			case logistic: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size_gpus[i], flt, logistic); break;
			case logistic_grad: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size_gpus[i], flt, logistic_grad); break;
			case rectified_linear: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i], A->size_gpus[i], flt, rectified_linear); break;
			case eq_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i],A->size_gpus[i], flt,eq_tensor); break;
			case lt_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i],A->size_gpus[i], flt,lt_tensor); break;
			case gt_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i],A->size_gpus[i], flt,gt_tensor); break;
			case ge_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i],A->size_gpus[i], flt,ge_tensor); break;
			case le_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i],A->size_gpus[i], flt,le_tensor); break;
			case ne_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i],A->size_gpus[i], flt,ne_tensor); break;
			case eq_scalar: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i],A->size_gpus[i], flt,eq_scalar); break;
			case lt_scalar: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i],A->size_gpus[i], flt,lt_scalar); break;
			case gt_scalar: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i],A->size_gpus[i], flt,gt_scalar); break;
			case ge_scalar: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i],A->size_gpus[i], flt,ge_scalar); break;
			case le_scalar: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i],A->size_gpus[i], flt,le_scalar); break;
			case ne_scalar: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i],A->size_gpus[i], flt,ne_scalar); break;
			case dropout_tensor: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, out->data_gpus[i],A->size_gpus[i], flt,dropout_tensor); break;
			case eq_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][0], A->shape_gpus[i][2], A->shape_gpus[i][3]*A->shape_gpus[i][2], eq_vec); break;
			case lt_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][0], A->shape_gpus[i][2], A->shape_gpus[i][3]*A->shape_gpus[i][2], lt_vec); break;
			case gt_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][0], A->shape_gpus[i][2], A->shape_gpus[i][3]*A->shape_gpus[i][2], gt_vec); break;
			case le_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][0], A->shape_gpus[i][2], A->shape_gpus[i][3]*A->shape_gpus[i][2], le_vec); break;
			case ge_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][0], A->shape_gpus[i][2], A->shape_gpus[i][3]*A->shape_gpus[i][2], ge_vec); break;
			case ne_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][0], A->shape_gpus[i][2], A->shape_gpus[i][3]*A->shape_gpus[i][2], ne_vec); break;
			case add_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][0], A->shape_gpus[i][2], A->shape_gpus[i][3]*A->shape_gpus[i][2], add_vec); break;
			case sub_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][0], A->shape_gpus[i][2], A->shape_gpus[i][3]*A->shape_gpus[i][2], sub_vec); break;
			case mul_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][0], A->shape_gpus[i][2], A->shape_gpus[i][3]*A->shape_gpus[i][2], mul_vec); break;
			case div_vec: kVectorWise<<<grid,THREADS_PER_BLOCKS>>>(A->data_gpus[i], B->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][0], A->shape_gpus[i][2], A->shape_gpus[i][3]*A->shape_gpus[i][2], div_vec); break;
			case print: kElementWise<<<block_size,THREADS_PER_BLOCKS>>>(A->data_gpus[i], NULL, NULL,A->size_gpus[i], flt,print); printf("\n"); break;

			default: throw "Unsupported operation!";
		}
		CUDA_CHECK_RETURN(hipPeekAtLastError());
		if(ops == print){ CUDA_CHECK_RETURN(hipDeviceSynchronize());}
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}


Tensor *softmax(Tensor *A){ Tensor *out = empty_like(A); softmax(A,out); return out; }
void softmax(Tensor *A, Tensor *out)
{
	dim3 grids(A->batches, A->maps);
	dim3 threads(A->rows > THREADS_PER_BLOCKS ? THREADS_PER_BLOCKS : A->rows, 1);
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kSoftMax<<<grids,threads >>>(A->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][2], A->shape_gpus[i][3]);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}

Tensor *argmax(Tensor *A){ Tensor *out = empty(A->batches,A->maps,A->rows,1); argmax(A,out); return out; }
void argmax(Tensor *A, Tensor *out)
{
	dim3 grids(A->batches, A->maps);
	dim3 threads(A->rows > THREADS_PER_BLOCKS ? THREADS_PER_BLOCKS : A->rows, 1);
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kArgmax<<<grids,threads >>>(A->data_gpus[i], out->data_gpus[i], A->shape_gpus[i][2], A->shape_gpus[i][3]);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}

void weightUpdate(Tensor *RMS, Tensor *grad, float RMS_multiplier, float learning_rate, int batch_size, weightUpdate_t strategy)
{

	int blocks = (RMS->size/THREADS_PER_BLOCKS) + 1;
	int gpus = 0;
	CUDA_CHECK_RETURN(hipGetDeviceCount(&gpus));
	for(int i = 0; i < gpus; i++)
	{
		CUDA_CHECK_RETURN(hipSetDevice(i));
		kWeightUpdate<<<blocks,THREADS_PER_BLOCKS>>>(RMS->data_gpus[i], grad->data_gpus[i], RMS_multiplier, learning_rate, batch_size, RMS->size, strategy);
		CUDA_CHECK_RETURN(hipPeekAtLastError());
	}
	CUDA_CHECK_RETURN(hipSetDevice(0));
}


float sum(Tensor *A)
{
	thrust::device_ptr<float> ptr(A->data);
	return thrust::reduce(ptr, ptr+A->size);
}

float max(Tensor *A)
{
	thrust::device_ptr<float> ptr(A->data);
	float res = -1.0f;
	return thrust::reduce(ptr, ptr+A->size,res, thrust::maximum<float>());
}

float min(Tensor *A)
{
	thrust::device_ptr<float> ptr(A->data);
	float res = -1.0f;
	return thrust::reduce(ptr, ptr+A->size,res, thrust::minimum<float>());
}


